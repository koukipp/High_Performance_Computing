/*
* This sample implements a separable convolution 
* of a 2D image with an arbitrary filter.
*/


#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <errno.h>
#include <string.h>
#include <time.h>
#include <sys/time.h>

unsigned int filter_radius;

#define FILTER_LENGTH 	(2 * filter_radius + 1)
#define ABS(val)  	((val)<0.0 ? (-(val)) : (val))
#define accuracy  	0.00005
#define CPU_EXEC

#define gpuErrchk(ans) { gpuAssert((ans), __LINE__, 1); }
inline void gpuAssert(hipError_t code, int line, int abort) {
	if (code != hipSuccess)  {
		fprintf(stderr,"%s, line %d\n", hipGetErrorString(code), line);
		if (abort) {
			hipDeviceReset();
			exit(code);
		}
	}
}

void *safeMalloc(size_t size, int line) {
    void *p = malloc(size);
    if (!p)
    {
        fprintf(stderr, "%s, line %d\n", strerror(errno), line);
        exit(EXIT_FAILURE);
    }
    return p;
}

////////////////////////////////////////////////////////////////////////////////
// Reference row convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionRowCPU(double *h_Dst, double *h_Src, double *h_Filter, 
	int imageW, int imageH, int filterR) {

	int x, y, k;
											
	for (y = 0; y < imageH; y++) {
		for (x = 0; x < imageW; x++) {
			double sum = 0;

			for (k = -filterR; k <= filterR; k++) {
				int d = x + k;

				if (d >= 0 && d < imageW) {
					sum += h_Src[y * imageW + d] * h_Filter[filterR - k];
				}     

				h_Dst[y * imageW + x] = sum;
			}
		}
	}			
}

////////////////////////////////////////////////////////////////////////////////
// Reference column convolution filter
////////////////////////////////////////////////////////////////////////////////
void convolutionColumnCPU(double *h_Dst, double *h_Src, double *h_Filter,
	int imageW, int imageH, int filterR) {

	int x, y, k;
	
	for (y = 0; y < imageH; y++) {
		for (x = 0; x < imageW; x++) {
			double sum = 0;

			for (k = -filterR; k <= filterR; k++) {
				int d = y + k;

				if (d >= 0 && d < imageH) {
					sum += h_Src[d * imageW + x] * h_Filter[filterR - k];
				}   
 
				h_Dst[y * imageW + x] = sum;
			}
		}
	}
}

__global__ void convolutionRowGPU(double *d_Dst, double *d_Src, double *d_Filter, 
	int filterR) {

	int k;
	double sum = 0.0;

	int padding_skip = filterR * (2 * filterR + blockDim.x * gridDim.x) + filterR;
	int idx_x = blockIdx.x * blockDim.x + threadIdx.x + padding_skip;
	int row_length = blockDim.x * gridDim.x + 2 * filterR;
	int elements_offset = (blockIdx.y * blockDim.y + threadIdx.y)* row_length;
	int dest = idx_x + elements_offset;

	for (k = -filterR; k <= filterR; k++) {
		sum += d_Src[dest + k] * d_Filter[filterR - k];
	}

	d_Dst[dest] = sum;
}

__global__ void convolutionColumnGPU(double *d_Dst, double *d_Src, double *d_Filter,
	int filterR) {
	
	int k;
	double sum = 0.0;

	int padding_skip = filterR * (2 * filterR + blockDim.x * gridDim.x) + filterR;
	int idx_y = blockIdx.y * blockDim.y + threadIdx.y;
	int row_length = blockDim.x * gridDim.x + 2 * filterR;
	int column_offset = blockIdx.x*blockDim.x + threadIdx.x + padding_skip;
	int dest = idx_y * blockDim.x * gridDim.x + blockIdx.x*blockDim.x + threadIdx.x;

	for (k = -filterR; k <= filterR; k++) {
		sum += d_Src[(idx_y + k) * row_length + column_offset] * d_Filter[filterR - k];
	}

	d_Dst[dest] = sum;
}

////////////////////////////////////////////////////////////////////////////////
// Main program
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv) {
	double
		*h_Filter,
		*h_Input,
		*h_Buffer,
		*h_OutputGPU;

	double
		*d_Filter,
		*d_Input,
		*d_Buffer,
		*d_OutputGPU;

	int imageW;
	int imageH;
	unsigned int i;

	if(argc==3) {
		filter_radius = (int)strtol(argv[1], (char **)NULL, 10);
		imageW = (int)strtol(argv[2], (char **)NULL, 10);
	}
	else {
		printf("Enter filter radius : ");
		if(scanf("%d", &filter_radius) != 1) {
			printf("Scanf Failed.\n");
		}

		printf("Enter image size. Should be a power of two and greater than %d : ",
			FILTER_LENGTH);
		if(scanf("%d", &imageW) != 1) {
			printf("Scanf Failed.\n");
		}
	}

	imageH = imageW;

	printf("Image Width x Height = %i x %i\n\n", imageW, imageH);
	printf("Allocating and initializing host arrays...\n");

	h_Filter    = (double *)safeMalloc(FILTER_LENGTH * sizeof(double), __LINE__);

	h_Input     = (double *)safeMalloc(imageW * imageH * sizeof(double), __LINE__);

	h_Buffer    = (double *)safeMalloc(imageW * imageH * sizeof(double), __LINE__);

	#ifdef CPU_EXEC
	double *h_OutputCPU;
	h_OutputCPU = (double *)safeMalloc(imageW * imageH * sizeof(double), __LINE__);
	#endif

	h_OutputGPU = (double *)safeMalloc(imageW * imageH * sizeof(double), __LINE__);

	gpuErrchk(hipMalloc( (void**) &d_Filter, FILTER_LENGTH * sizeof(double)));

	gpuErrchk(hipMalloc( (void**) &d_Input, (imageW + 2 * filter_radius)
		* (imageH + 2 * filter_radius) * sizeof(double)));

	gpuErrchk(hipMalloc( (void**) &d_Buffer, (imageW + 2 * filter_radius)
		* (imageH + 2 * filter_radius) * sizeof(double)));

	gpuErrchk(hipMalloc( (void**) &d_OutputGPU, imageW * imageH * sizeof(double)));

	srand(200);

	for (i = 0; i < FILTER_LENGTH; i++) {
		h_Filter[i] = (double)(rand() % 16);
	}

	for (i = 0; i < imageW * imageH; i++) {
		h_Input[i] = (double)rand() / ((double)RAND_MAX / 255) + 
					(double)rand() / (double)RAND_MAX;
	}

	printf("CPU computation...\n");

#ifdef CPU_EXEC
    struct timespec start, end; 
	float time_taken; 
    clock_gettime(CLOCK_MONOTONIC, &start); 

	convolutionRowCPU(h_Buffer, h_Input, h_Filter, imageW, imageH,
		filter_radius);
	convolutionColumnCPU(h_OutputCPU, h_Buffer, h_Filter, imageW, imageH, 
		filter_radius);

    clock_gettime(CLOCK_MONOTONIC, &end); 
    time_taken = (end.tv_sec - start.tv_sec) * 1e9; 
    time_taken = (time_taken + (end.tv_nsec - start.tv_nsec)) * 1e-6; 
#endif

	dim3 grid;
	if(imageW < 32){
		grid.x = 1;
		grid.y = 1;
	}
	else{
		grid.x = imageW/32;
		grid.y = imageH/32;
	}
	
	dim3 block;
	block.x = imageW/grid.x;
	block.y = imageH/grid.y;

	hipEvent_t start_event, stop_event;
	hipEventCreate(&start_event);
	hipEventCreate(&stop_event);

	h_Buffer = (double *)realloc(h_Buffer, (imageW + 2 * filter_radius) 
		* (imageH + 2 * filter_radius) * sizeof(double));
	if (!h_Input) {
		fprintf(stderr, "%s\n", strerror(errno));
		exit(EXIT_FAILURE);
	}

	memset(h_Buffer, 0, (imageW + 2 * filter_radius) 
		* (imageH + 2 * filter_radius) * sizeof(double));

	gpuErrchk(hipMemset( (void*) d_Buffer, 0, (imageW + 2 * filter_radius)
		* (imageH + 2 * filter_radius) * sizeof(double)));

	for(i=0; i<imageH; i++) {
		memcpy(&h_Buffer[filter_radius * (2 * filter_radius + imageW) + 
			filter_radius + i * (2 * filter_radius + imageW)], &h_Input[i * imageW],
			imageW * sizeof(double));
	}

	hipEventRecord(start_event);
	
	gpuErrchk(hipMemcpy(d_Filter, h_Filter, FILTER_LENGTH * sizeof(double), 
		hipMemcpyHostToDevice));
	gpuErrchk(hipMemcpy(d_Input, h_Buffer, (imageW + 2 * filter_radius) 
		* (imageH + 2 * filter_radius) * sizeof(double), hipMemcpyHostToDevice));

	convolutionRowGPU<<<grid, block>>>(d_Buffer, d_Input, d_Filter,
		filter_radius);

	convolutionColumnGPU<<<grid, block>>>(d_OutputGPU, d_Buffer, d_Filter,
		filter_radius);

	gpuErrchk(hipPeekAtLastError());
	gpuErrchk(hipMemcpy(h_OutputGPU, d_OutputGPU, imageW * imageH * sizeof(double), 
		hipMemcpyDeviceToHost));

	hipEventRecord(stop_event);
	hipEventSynchronize(stop_event);

	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start_event, stop_event);
    printf("%f\n",milliseconds);
    #ifdef CPU_EXEC
    printf("%f\n", time_taken);
	double max = ABS(h_OutputGPU[0] - h_OutputCPU[0]);
	for(i = 0; i < imageW * imageH; i++) {
		if(max < ABS(h_OutputGPU[i] - h_OutputCPU[i]))
			max = ABS(h_OutputGPU[i] - h_OutputCPU[i]);
	}
	fprintf(stderr, "Max observed error: %f\n", max);

	for(i = 0; i < imageW * imageH; i++) {
		if(ABS(h_OutputGPU[i] - h_OutputCPU[i]) > accuracy) {
			fprintf(stderr, "Images differ\n");
			break;
		}
	}
    #endif

	// free all the allocated memory

	free(h_Buffer);
	free(h_Input);
	free(h_Filter);
	free(h_OutputGPU);
	#ifdef CPU_EXEC
	free(h_OutputCPU);
	#endif

	hipFree(d_OutputGPU);
	hipFree(d_Buffer);
	hipFree(d_Input);
	hipFree(d_Filter);

	hipDeviceReset();

	return 0;
}
